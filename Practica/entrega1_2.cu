#include "hip/hip_runtime.h"
/*Realizar un programa CUDA que dado un vector V de N números enteros multiplique a 
cada número por una constante C, se deben realizar dos implementaciones:
a.Tanto C como N deben ser pasados como parámetros al kernel.
b.Tanto C como N deben estar almacenados en la memoria de constantes de la GPU*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
//M and N number of threads (grid and block)




     
__global__ void addAll( const int array[] , int dim,float result[], const int thread_number)
{
    int index = blockIdx.x* blockDim.x* blockDim.y* blockDim.z+threadIdx.z* blockDim.y* blockDim.x+ threadIdx.y* blockDim.x+ threadIdx.x;
 	//printf("sum:%i\n",  result[0]);
	if(index<dim){
		if(dim<=thread_number){ //if more threads than array size
		//	printf("Thread %i; Adding value of index %i\n", index, index, array[index]);
			atomicAdd(result,array[index]);
		}
		else{ //if less threads than array size
			if(index!=thread_number-1){//if not last thread deal with size_array/thread_nb array entries
				for(int i=index*(int)(dim/thread_number); i< index*(int)(dim/thread_number)+(int)(dim/thread_number); i++){
			//		printf("Thread %i; Adding value of index %i\n", index, i, array[i]);
					atomicAdd(result,array[i]);
				}
			}
			else{ //if last thread deal with all remaining array entries
				for(int i=index*(int)(dim/thread_number); i< dim; i++){
				//	printf("Thread %i; Adding value of index %i\n",index, i, array[i]);
					atomicAdd(result,array[i]);
				}
			}
		}
		//printf("sum:%i\n",  result[0]);
	}
	
} 
__global__ void sigma( const int array[] , int dim,float result[], const float mean, const int thread_number)
{
	 int index = blockIdx.x* blockDim.x* blockDim.y* blockDim.z+threadIdx.z* blockDim.y* blockDim.x+ threadIdx.y* blockDim.x+ threadIdx.x;
 	//printf("sum:%i\n",  result[0]);
	if(index<dim){
		if(dim<=thread_number){ //if more threads than array size
			//printf("Thread %i; Adding value of index %i\n", index, index, array[index]);
			atomicAdd(result,(array[index]-mean)*(array[index]-mean));
		}
		else{ //if less threads than array size
			if(index!=thread_number-1){//if not last thread deal with size_array/thread_nb array entries
				for(int i=index*(int)(dim/thread_number); i< index*(int)(dim/thread_number)+(int)(dim/thread_number); i++){
					//printf("Thread %i; Adding value of index %i\n", index, i, array[i]);
					atomicAdd(result,(array[i]-mean)*(array[i]-mean));
				}
			}
			else{ //if last thread deal with all remaining array entries
				for(int i=index*(int)(dim/thread_number); i< dim; i++){
					//printf("Thread %i; Adding value of index %i\n",index, i, array[i]);
					atomicAdd(result,(array[i]-mean)*(array[i]-mean));
				}
			}
		}
		//printf("sum:%i\n",  result[0]);
	}


}


    
int main(int argc, char *argv[]){
	//Measure time
	clock_t time_begin;
	time_begin=clock();
    // pointers to host & device arrays
     int *device_array = 0;
     int *host_array = 0;
	 int size_array=9;
	 float *d_sum=NULL;
	 float *h_sum= 0;
	 float mean;
	 float final_res;
	 int M=1, N=1;
	 if(argc == 4){
		 size_array=atoi(argv[1]);
		 N=atoi(argv[2]);
		 M=atoi(argv[3]);
	 }
	 h_sum=( float*)malloc(sizeof( float));
	 h_sum[0]=0;
      // malloc a host array
     host_array = (int*)malloc( size_array * sizeof(int));
	
    for(int i=0; i<size_array; i++){
        host_array[i]=rand()%10;
       // printf("%i\t", host_array[i]);
    }
    printf("\n");
	
	
     // hipMalloc a device array
     hipMalloc(&device_array,size_array * sizeof(int));    
	 hipError_t er=hipMalloc(&d_sum, sizeof(float));  
    // download and inspect the result on the host:
    hipError_t e=hipMemcpy(device_array, host_array, sizeof(int)*size_array, hipMemcpyHostToDevice); 
	hipError_t error=hipMemcpy(d_sum, h_sum, sizeof(int), hipMemcpyHostToDevice);
	//cudaerrorinvalidvalue(11)

    dim3 bloque(N,N); //Bloque bidimensional de N*N hilos
    dim3 grid(M,M);  //Grid bidimensional de M*M bloques
	int thread_number= N*N*M*M;
    addAll<<<grid, bloque>>>(device_array, size_array , d_sum, thread_number);
    hipDeviceSynchronize();
    // download and inspect the result on the host:
   //hipMemcpy(host_array, device_array, sizeof(int)*size_array, hipMemcpyDeviceToHost); 
	hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost); 
   
    printf("Sum of array: %f\n", h_sum[0]);
	mean=h_sum[0]/size_array;
	h_sum[0]=0;

	hipMemcpy(d_sum, h_sum, sizeof(int), hipMemcpyHostToDevice);
	sigma<<<grid, bloque>>>(device_array, size_array , d_sum, mean, thread_number);
	hipDeviceSynchronize();
	hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost); 

	 printf("Sigma: %f\n", h_sum[0]);
	
	final_res = sqrt(h_sum[0]/(size_array-1));
	
	printf("Final result: %f\n", final_res);
     // deallocate memory
      free(host_array);free(h_sum);
      hipFree(device_array); hipFree(d_sum);

	  printf("Time elapsed: %f seconds\n", (((float)clock() - (float)time_begin) / 1000000.0F ) * 1000  ); //1.215s

}