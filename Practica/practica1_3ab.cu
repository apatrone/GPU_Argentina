#include "hip/hip_runtime.h"
/*Realizar un programa CUDA que dado un vector V de N números enteros multiplique a 
cada número por una constante C, se deben realizar dos implementaciones:
a.Tanto C como N deben ser pasados como parámetros al kernel.
b.Tanto C como N deben estar almacenados en la memoria de constantes de la GPU*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//M and N number of threads (grid and block)
#define M 1 
#define N 1   



     
__global__ void multiply( const int a[] ,const int b[], int c[] , const int dim,const int thread_number)
{
    int index = blockIdx.x* blockDim.x* blockDim.y* blockDim.z+threadIdx.z* blockDim.y* blockDim.x+ threadIdx.y* blockDim.x+ threadIdx.x;
 	
	if(index<dim){
		if(dim<=thread_number){ //if more threads than array size
			printf("Thread %i; Modifying value of index %i\n ", index, index);
			c[index]=a[index]+b[index]; 
			
		}
		else{ //if less threads than array size
			if(index!=thread_number-1){//if not last thread deal with size_array/thread_nb array entries
				for(int i=index*(int)(dim/thread_number); i< index*(int)(dim/thread_number)+(int)(dim/thread_number); i++){
					printf("Thread %i; Modifying value of index %i \n", index, i);
					c[i]=a[i]+b[i]; 
				}
			}
			else{ //if last thread deal with all remaining array entries
				for(int i=index*(int)(dim/thread_number); i< dim; i++){
					printf("Thread %i; Modifying value of index %i\n",index, i );
					c[i]=a[i]+b[i]; 
				}
			}
		}
	}
	
} 

    
int main(int argc, char *argv[]){
	//Measure time
	clock_t time_begin;
	time_begin=clock();
    // pointers to host & device arrays
      int **d_array1 = 0,**d_array2 = 0,**d_array3 = 0;
      int **h_array1 = 0,**h_array2 = 0,**h_array3 = 0;
	  int size_array=10; //here, size_array =L where each matrix = L * L
      // malloc columns of host arrays
      h_array1 = (int*)malloc( size_array * sizeof(int*));
	  h_array2 = (int*)malloc( size_array * sizeof(int*));
	  h_array3 = (int*)malloc( size_array * sizeof(int*));
	  //malloc rows of host arrays
	  for(int i=0; i<size_array; i++){
		  h_array1[i]=(int*)malloc( size_array * sizeof(int));
		  h_array2[i]=(int*)malloc( size_array * sizeof(int));
		  h_array3[i]=(int*)malloc( size_array * sizeof(int));
	  }
    for(int i=0; i<size_array; i++){
		for(int j=0; j<size_array; j++){
			h_array1[i][j]=rand()%10;
			h_array2[i][j]=rand()%10;
			printf("%i|%i\t",  h_array1[i][j], h_array2[i][j]);
		}
		printf("\n");
    }

     // hipMalloc a device array
    hipMalloc(&d_array1,size_array * sizeof(int));    
	hipMalloc(&d_array2,size_array * sizeof(int));  
	hipMalloc(&d_array3,size_array * sizeof(int));  
    // download and inspect the result on the host:
    hipMemcpy(d_array1, h_array1, sizeof(int)*size_array, hipMemcpyHostToDevice);   
	hipMemcpy(d_array2, h_array2, sizeof(int)*size_array, hipMemcpyHostToDevice);   

    dim3 bloque(N,N); //Bloque bidimensional de N*N hilos
    dim3 grid(M,M);  //Grid bidimensional de M*M bloques
	int thread_number= N*N*M*M;
    multiply<<<grid, bloque>>>(d_array1, d_array2 , d_array3,size_array, thread_number);
    hipDeviceSynchronize();
    // download and inspect the result on the host:
    hipMemcpy(h_array3, d_array3, sizeof(int)*size_array, hipMemcpyDeviceToHost); 

    for(int i=0; i<size_array; i++)
        printf("%i\t", h_array3[i]);

	
     // deallocate memory
      free(h_array3); free(h_array2); free(h_array1);
      hipFree(d_array3);hipFree(d_array2);hipFree(d_array1);

	  printf("Time elapsed: %f seconds\n", (((float)clock() - (float)time_begin) / 1000000.0F ) * 1000  ); //1.215s

}