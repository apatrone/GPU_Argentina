#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
//M and N number of threads (grid and block)
#define M 1 
#define N 2


     
__global__ void multiply( const int a[] ,const int b[], int c[] , const int sqrt_dim,const int thread_number)
{
    int index = blockIdx.x* blockDim.x* blockDim.y* blockDim.z+threadIdx.z* blockDim.y* blockDim.x+ threadIdx.y* blockDim.x+ threadIdx.x;
 	//for an element in matrix[i][j] , its coordinate k in array[] is i+j*sqrt(size_array)
	int index_i = index < sqrt_dim ? index : (int)index%sqrt_dim; 
   	int index_j = (index-index_i)/sqrt_dim;
	int dim=sqrt_dim*sqrt_dim;


	if(index<dim){
		c[index]=0;
		if(dim<=thread_number){ //if more threads than array size
			printf("Thread %i; Modifying value of index %i\n ", index, index);
			for(int i=0; i<sqrt_dim;i++){ //row of first matrix
				c[index]+=a[i+index_j * sqrt_dim ]*b[index_i+ i* sqrt_dim]; 
				//printf("c[%i]+=a[%i]*b[%i]\n", index,i+index_j * sqrt_dim ,index_i+ i* sqrt_dim  );
			}
			
		}
		else{ //if less threads than array size
				
				if(index!=thread_number-1){//if not last thread deal with size_array/thread_nb array entries
					for(int i=index*(int)(dim/thread_number); i< index*(int)(dim/thread_number)+(int)(dim/thread_number); i++){
						printf("Thread %i; Modifying value of index %i \n", index, i);
						index_i =  (int)i%sqrt_dim; 
						index_j = (i-index_i)/sqrt_dim;
						for(int j=0; j<sqrt_dim;j++){ //row of first matrix
							c[i]+=a[j+index_j * sqrt_dim ]*b[index_i+ j* sqrt_dim]; 
						} 
					}
				}
				else{ //if last thread deal with all remaining array entries
					for(int i=index*(int)(dim/thread_number); i< dim; i++){
						printf("Thread %i; Modifying value of index %i\n",index, i );
						index_i = (int)i%sqrt_dim; 
						index_j = (i-index_i)/sqrt_dim;
						for(int j=0;j<sqrt_dim;j++){ //row of first matrix
							c[i]+=a[j+index_j * sqrt_dim ]*b[index_i+ j* sqrt_dim]; 
						}
					}
				}
			}
		}
	
} 

    
int main(int argc, char *argv[]){
	//Measure time
	clock_t time_begin;
	time_begin=clock();
    // pointers to host & device arrays
      int *d_array1 = 0,*d_array2 = 0,*d_array3 = 0;
      int *h_array1 = 0,*h_array2 = 0,*h_array3 = 0;
	  int size_array=9; //here, size_array =L hqs to be a square
      // malloc columns of host arrays
      h_array1 = (int*)malloc( size_array * sizeof(int));
	h_array2 = (int*)malloc( size_array * sizeof(int));
	h_array3 = (int*)malloc( size_array * sizeof(int));
	  
		  
 
	for(int i=0; i<size_array; i++){
		h_array1[i]=rand()%10;
		h_array2[i]=rand()%10;
		printf("%i|%i\t",  h_array1[i], h_array2[i]);
		if((i+1)%(int)sqrt((float)size_array)==0)
			printf("\n");
	}
	printf("\n");
 

     // hipMalloc a device array
    hipMalloc(&d_array1,size_array * sizeof(int));    
	hipMalloc(&d_array2,size_array * sizeof(int));  
	hipMalloc(&d_array3,size_array * sizeof(int));  
    // download and inspect the result on the host:
    hipMemcpy(d_array1, h_array1, sizeof(int)*size_array, hipMemcpyHostToDevice);   
	hipMemcpy(d_array2, h_array2, sizeof(int)*size_array, hipMemcpyHostToDevice);   

    dim3 bloque(N,N); //Bloque bidimensional de N*N hilos
    dim3 grid(M,M);  //Grid bidimensional de M*M bloques
	int thread_number= N*N*M*M;
    multiply<<<grid, bloque>>>(d_array1, d_array2 , d_array3,sqrt((float)size_array), thread_number);
    hipDeviceSynchronize();
    // download and inspect the result on the host:
    hipMemcpy(h_array3, d_array3, sizeof(int)*size_array, hipMemcpyDeviceToHost); 

    for(int i=0; i<size_array; i++){
        printf("%i\t", h_array3[i]);
	if((i+1)%(int)(sqrt((float)size_array))==0)
		printf("\n");
	}
	printf("\n");	
     // deallocate memory
      free(h_array3); free(h_array2); free(h_array1);
      hipFree(d_array3);hipFree(d_array2);hipFree(d_array1);

	  printf("Time elapsed: %f seconds\n", (((float)clock() - (float)time_begin) / 1000000.0F ) * 1000  ); //1.18s

}
