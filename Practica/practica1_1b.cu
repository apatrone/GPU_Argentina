#include "hip/hip_runtime.h"
/*Realizar un programa CUDA que dado un vector V de N números enteros multiplique a 
cada número por una constante C, se deben realizar dos implementaciones:
a.Tanto C como N deben ser pasados como parámetros al kernel.
b.Tanto C como N deben estar almacenados en la memoria de constantes de la GPU*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//M and N number of threads (grid and block)
#define M 1 
#define N 1   

int h_array_size=10; //host 
__constant__ int d_array_size; //device
int h_c=2; //host 
__constant__ int d_c; //device

     
__global__ void multiply( int array[] , const int thread_number)
{
    int index = blockIdx.x* blockDim.x* blockDim.y* blockDim.z+threadIdx.z* blockDim.y* blockDim.x+ threadIdx.y* blockDim.x+ threadIdx.x;
 	
	if(index<d_array_size){
		if(d_array_size<=thread_number){ //if more threads than array size
			printf("Thread %i; Modifying value of index %i for %i * %i because < d_array_size %i\n", index, index, array[index], d_c, d_array_size);
			array[index]*=d_c; 
		}
		else{ //if less threads than array size
			if(index!=thread_number-1){//if not last thread deal with h_array_size/thread_nb array entries
				for(int i=index*(int)(d_array_size/thread_number); i< index*(int)(d_array_size/thread_number)+(int)(d_array_size/thread_number); i++){
					printf("Thread %i; Modifying value of index %i for %i * %i because < d_array_size %i\n", index, i, array[i], d_c, d_array_size);
					array[i]*=d_c; 
				}
			}
			else{ //if last thread deal with all remaining array entries
				for(int i=index*(int)(d_array_size/thread_number); i< d_array_size; i++){
					printf("Thread %i; Modifying value of index %i for %i * %i because < d_array_size %i\n",index, i, array[i], d_c, d_array_size);
					array[i]*=d_c; 
				}
			}
		}
	}
	
} 

    
int main(int argc, char *argv[]){
	//Measure time
	clock_t time_begin;
	time_begin=clock();
    // pointers to host & device arrays
      int *device_array = 0;
      int *host_array = 0;
	  int h_array_size=10;
      
	  //copy variables from host to device
	  hipMemcpyToSymbol(HIP_SYMBOL(d_array_size),&h_array_size,sizeof(h_array_size));
	  hipMemcpyToSymbol(HIP_SYMBOL(d_c),&h_c,sizeof(h_c));
	  // malloc a host array
      host_array = (int*)malloc( h_array_size * sizeof(int));

    for(int i=0; i<h_array_size; i++){
        host_array[i]=rand()%10;
        printf("%i\t", host_array[i]);
    }
    printf("\n");

    // hipMalloc a device array
    hipMalloc(&device_array,h_array_size * sizeof(int));    
    // download and inspect the result on the host:
    hipMemcpy(device_array, host_array, sizeof(int)*h_array_size, hipMemcpyHostToDevice);         

    dim3 bloque(N,N); //Bloque bidimensional de N*N hilos
    dim3 grid(M,M);  //Grid bidimensional de M*M bloques
	int thread_number= N*N*M*M;
    multiply<<<grid, bloque>>>(device_array, thread_number);
    hipDeviceSynchronize();
    // download and inspect the result on the host:
    hipMemcpy(host_array, device_array, sizeof(int)*h_array_size, hipMemcpyDeviceToHost); 

    for(int i=0; i<h_array_size; i++)
        printf("%i\t", host_array[i]);


     // deallocate memory
      free(host_array);
      hipFree(device_array);
	
	 printf("Time elapsed: %f seconds\n", (((float)clock() - (float)time_begin) / 1000000.0F ) * 1000  ); //1.167s

}