#include "hip/hip_runtime.h"
/*Realizar un programa CUDA que dado un vector V de N números enteros multiplique a 
cada número por una constante C, se deben realizar dos implementaciones:
a.Tanto C como N deben ser pasados como parámetros al kernel.
b.Tanto C como N deben estar almacenados en la memoria de constantes de la GPU*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//M and N number of threads (grid and block)
#define M 1 
#define N 1   



     
__global__ void multiply( int array[] , int dim,  const int c, const int thread_number)
{
    int index = blockIdx.x* blockDim.x* blockDim.y* blockDim.z+threadIdx.z* blockDim.y* blockDim.x+ threadIdx.y* blockDim.x+ threadIdx.x;
 	
	if(index<dim){
		if(dim<=thread_number){ //if more threads than array size
			printf("Thread %i; Modifying value of index %i for %i * %i because < dim %i\n", index, index, array[index], c, dim);
			array[index]*=c; 
		}
		else{ //if less threads than array size
			if(index!=thread_number-1){//if not last thread deal with size_array/thread_nb array entries
				for(int i=index*(int)(dim/thread_number); i< index*(int)(dim/thread_number)+(int)(dim/thread_number); i++){
					printf("Thread %i; Modifying value of index %i for %i * %i because < dim %i\n", index, i, array[i], c, dim);
					array[i]*=c; 
				}
			}
			else{ //if last thread deal with all remaining array entries
				for(int i=index*(int)(dim/thread_number); i< dim; i++){
					printf("Thread %i; Modifying value of index %i for %i * %i because < dim %i\n",index, i, array[i], c, dim);
					array[i]*=c; 
				}
			}
		}
	}
	
} 

    
int main(int argc, char *argv[]){
	//Measure time
	clock_t time_begin;
	time_begin=clock();
    // pointers to host & device arrays
      int *device_array = 0;
      int *host_array = 0;
	  int size_array=10;
      // malloc a host array
      host_array = (int*)malloc( size_array * sizeof(int));

    for(int i=0; i<size_array; i++){
        host_array[i]=rand()%10;
        printf("%i\t", host_array[i]);
    }
    printf("\n");

      // hipMalloc a device array
      hipMalloc(&device_array,size_array * sizeof(int));    
    // download and inspect the result on the host:
    hipMemcpy(device_array, host_array, sizeof(int)*size_array, hipMemcpyHostToDevice);         

    dim3 bloque(N,N); //Bloque bidimensional de N*N hilos
    dim3 grid(M,M);  //Grid bidimensional de M*M bloques
	int thread_number= N*N*M*M;
    multiply<<<grid, bloque>>>(device_array, size_array , 2, thread_number);
    hipDeviceSynchronize();
    // download and inspect the result on the host:
    hipMemcpy(host_array, device_array, sizeof(int)*size_array, hipMemcpyDeviceToHost); 

    for(int i=0; i<size_array; i++)
        printf("%i\t", host_array[i]);

	
     // deallocate memory
      free(host_array);
      hipFree(device_array);

	  printf("Time elapsed: %f seconds\n", (((float)clock() - (float)time_begin) / 1000000.0F ) * 1000  ); //1.215s

}