#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

void secuential(const int a[] ,const int b[], int c[], const unsigned int sqrt_dim);
     
__global__ void multiply(  const int* A, const int* B,int* C, int width, int tile_width)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	
    // Index of the first sub-matrix of A processed by the block
    int aBegin = width * tile_width * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + width - 1;
		
    // Step size used to iterate through the sub-matrices of A
    int aStep  = tile_width;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = tile_width * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = tile_width * width;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;
	

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

		extern __shared__ int shared[];

		int *As=&shared[0];
		int *Bs=&shared[tile_width*tile_width];

        As[ty+tile_width*tx] = A[a + width * ty + tx];
        Bs[ty+tile_width*tx] = B[b + width * ty + tx];
        __syncthreads();

		//Csub=B[b + width * ty + tx] + A[a +  ty + width*tx];
        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < tile_width; ++k)
        {
            //Csub += As[ty+tile_width*k] * Bs[k+tile_width*tx];//a*b
			Csub += As[ty+tile_width*k] * Bs[tx+tile_width*k]; //a*b^t
        }
		
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = width * tile_width * by + tile_width * bx;
    C[c + width * ty + tx] = Csub;
	C[c + width * ty + tx]+=B[c + width * ty + tx] + A[c + width * tx + ty];

}

void constantInit(int *data, int size, int val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }

}

void randomInit(int* data, int size)
{
   for (int i = 0; i < size; ++i)
   data[i] = rand()%10;//rand() / (float)RAND_MAX;
}

 
int main(int argc, char** argv)
{
	clock_t time_begin;
	unsigned int size_array  = (argc > 1)? atoi (argv[1]): 1024;
	unsigned int tile_width = (argc > 2)? atoi (argv[2]): 2;	
	bool verbose= (argc>3)? (argv[3][0]=='v'): false;

   int* h_array1 = (int*) malloc(sizeof(int) * size_array*size_array); 
   int* h_array2 = (int*) malloc(sizeof(int) * size_array*size_array);
   int* h_array3 = (int*) malloc(sizeof(int) * size_array*size_array);
   int* h_array_sec = (int*) malloc(sizeof(int) * size_array*size_array);
 

   constantInit(h_array1, size_array*size_array,1);
   constantInit(h_array2, size_array*size_array,1);
    
	if(verbose){
		printf("A:\n");
		for(int i=0; i<size_array*size_array; i++){
			printf("%i\t", h_array1[i]); 
			if((i+1)%size_array==0) printf("\n");
		}
		printf("\n");
		printf("B:\n");
		for(int i=0; i<size_array*size_array; i++){
			printf("%i\t", h_array2[i]);
			if((i+1)%size_array==0) printf("\n");
		}
		printf("\n");
	}
  
   int *d_array1,*d_array2, *d_array3;
   hipMalloc((void**) &d_array1, sizeof(int) * size_array*size_array);
   hipMalloc((void**) &d_array2, sizeof(int) * size_array*size_array);
   hipMalloc((void**) &d_array3, sizeof(int) * size_array*size_array);
   hipMemcpy(d_array1, h_array1, sizeof(int) * size_array*size_array, hipMemcpyHostToDevice);
   hipMemcpy(d_array2, h_array2, sizeof(int) * size_array*size_array, hipMemcpyHostToDevice);

   dim3 bloque(tile_width, tile_width);
   dim3 grid(size_array / bloque.x, size_array / bloque.y);
   time_begin=clock();
   multiply<<< grid, bloque, tile_width*tile_width*tile_width*tile_width >>>( d_array1, d_array2,d_array3, size_array, tile_width);
   hipMemcpy(h_array3, d_array3, sizeof(int) * size_array*size_array, hipMemcpyDeviceToHost);
   printf("GPU time: %f seconds\n", (((float)clock() - (float)time_begin) / 1000000.0F ) * 1000  ); 

   if(verbose){
		printf("Array C=B + AB^t + A^t :\n");
		for(int i=0; i<size_array*size_array; i++){
			printf("%i\t", h_array3[i]);
			if((i+1)%size_array==0) printf("\n");
		}
	}
	time_begin=clock();
	secuential(h_array1, h_array2, h_array_sec, size_array);
	printf("CPU time: %f seconds\n", (((float)clock() - (float)time_begin) / 1000000.0F ) * 1000  ); 

	bool b=true;
	for(int i=0; i<size_array; i++){
		if(h_array_sec[i] !=  h_array3[i]){
			printf("GPU and CPU have different results (at least) at position %i\n", i);
			b=false;
			break;		
		}
	}
	if(b)
		printf("GPU and CPU have the same results\n");
   free(h_array1);
   free(h_array2);
   free(h_array3);
   hipFree(d_array1);
   hipFree(d_array2);
   hipFree(d_array3);
 
}

void secuential(const int a[] ,const int b[], int c[], const unsigned int sqrt_dim){
	int dim = sqrt_dim* sqrt_dim;
	int index_i, index_j;
	//int *c= (int *)malloc ( dim * sizeof(int));
	for(int i=0; i< dim; i++){
		index_i = (int)i%sqrt_dim; 
		index_j = (i-index_i)/sqrt_dim;
		c[i]= b[i]; //c= b
		c[i]+= a[index_j+ index_i * sqrt_dim]; //c+= a^t
		for(int j=0;j<sqrt_dim;j++){ //row of first matrix
			c[i]+=a[j+index_j * sqrt_dim ]*b[j + index_i*sqrt_dim]; //c+= a*b^t
		}
	}

	/*printf("Sequential result: Array C=B + AB^t + A^t :\n");
    for(int i=0; i<dim; i++){
        printf("%i\t", c[i]);
		if((i+1)%(int)(sqrt((float)dim))==0)
			printf("\n");
	}
	printf("\n");*/
	//free(c);
}
